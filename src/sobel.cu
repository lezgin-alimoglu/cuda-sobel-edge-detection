#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>


using namespace cv;
using namespace std;

int main(int argc, char** argv) {
    VideoCapture cap;
    bool use_camera = true;
    bool save_video = false;

    if (argc > 1) {
        cap.open(argv[1]);
        use_camera = false;
        if (argc > 2 && string(argv[2]) == "save") {
            save_video = true;
        }
    } else {
        cap.open(0);
    }

    if (!cap.isOpened()) {
        cerr << "Error: Could not open video source." << endl;
        return -1;
    }

    int frame_width = static_cast<int>(cap.get(CAP_PROP_FRAME_WIDTH));
    int frame_height = static_cast<int>(cap.get(CAP_PROP_FRAME_HEIGHT));
    int fps = static_cast<int>(cap.get(CAP_PROP_FPS));
    if (fps == 0) fps = 35;

    VideoWriter writer;
    if (save_video) {
        writer.open("output.avi", VideoWriter::fourcc('M','J','P','G'), fps, Size(frame_width, frame_height), false);
        if (!writer.isOpened()) {
            cerr << "Error: Could not open the video writer." << endl;
            return -1;
        }
        cout << "Video recording enabled, saving to output.avi" << endl;
    }

    Ptr<cuda::Filter> sobel_x = cuda::createSobelFilter(CV_8UC1, CV_16S, 1, 0, 3);
    Ptr<cuda::Filter> sobel_y = cuda::createSobelFilter(CV_8UC1, CV_16S, 0, 1, 3);

    Mat frame;
    cuda::GpuMat d_frame, d_gray, d_grad_x, d_grad_y, d_abs_grad_x, d_abs_grad_y, d_grad;

    cout << "Press 'q' to quit." << endl;

    while (true) {
        cap >> frame;
        if (frame.empty()) break;

        d_frame.upload(frame);
        cuda::cvtColor(d_frame, d_gray, COLOR_BGR2GRAY);

        sobel_x->apply(d_gray, d_grad_x);
        sobel_y->apply(d_gray, d_grad_y);

        cuda::abs(d_grad_x, d_abs_grad_x);
        cuda::abs(d_grad_y, d_abs_grad_y);

        cuda::addWeighted(d_abs_grad_x, 0.5, d_abs_grad_y, 0.5, 0, d_grad);

        cuda::GpuMat d_grad_8u;
        d_grad.convertTo(d_grad_8u, CV_8U);

        Mat grad;
        d_grad_8u.download(grad);

        imshow("Original", frame);
        imshow("CUDA Sobel", grad);

        if (save_video) {
            writer.write(grad);
        }

        if ((char)waitKey(1) == 'q') break;
    }

    cap.release();
    if (save_video) writer.release();
    destroyAllWindows();

    return 0;
}
