#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace cv;
using namespace std;

// CUDA kernel for Sobel edge detection
__global__ void sobelKernel(const unsigned char* input, unsigned char* output, 
                           int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Sobel kernels
    const int sobelX[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    const int sobelY[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
    
    int gx = 0, gy = 0;
    
    // Apply Sobel kernels
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int nx = x + i;
            int ny = y + j;
            
            // Handle boundary conditions
            if (nx < 0) nx = 0;
            if (nx >= width) nx = width - 1;
            if (ny < 0) ny = 0;
            if (ny >= height) ny = height - 1;
            
            int pixel = input[ny * width + nx];
            
            gx += pixel * sobelX[i + 1][j + 1];
            gy += pixel * sobelY[i + 1][j + 1];
        }
    }
    
    // Calculate magnitude
    int magnitude = (int)sqrtf((float)(gx * gx + gy * gy));
    
    // Clamp to 0-255 range
    magnitude = min(255, max(0, magnitude));
    
    output[y * width + x] = (unsigned char)magnitude;
}

// CUDA kernel for color to grayscale conversion
__global__ void rgbToGrayKernel(const unsigned char* input, unsigned char* output, 
                                int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = (y * width + x) * 3; // 3 channels (BGR)
    
    // BGR to grayscale conversion
    unsigned char b = input[idx];
    unsigned char g = input[idx + 1];
    unsigned char r = input[idx + 2];
    
    // Use standard luminance formula
    unsigned char gray = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
    
    output[y * width + x] = gray;
}

// Host function to process image with CUDA
extern "C" void processImageWithCUDA(const Mat& input, Mat& output) {
    int width = input.cols;
    int height = input.rows;
    
    // Allocate device memory
    unsigned char *d_input, *d_gray, *d_output;
    size_t inputSize = width * height * 3 * sizeof(unsigned char); // BGR
    size_t graySize = width * height * sizeof(unsigned char);
    
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_gray, graySize);
    hipMalloc(&d_output, graySize);
    
    // Copy input to device 
    hipMemcpy(d_input, input.data, inputSize, hipMemcpyHostToDevice);
    
    // Define block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);
    
    // Convert to grayscale
    rgbToGrayKernel<<<gridSize, blockSize>>>(d_input, d_gray, width, height);
    
    // Apply Sobel filter
    sobelKernel<<<gridSize, blockSize>>>(d_gray, d_output, width, height);
    
    // Copy result back to host
    output = Mat(height, width, CV_8UC1);
    hipMemcpy(output.data, d_output, graySize, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_output);
    
    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cerr << "CUDA error: " << hipGetErrorString(error) << endl;
    }
} 


